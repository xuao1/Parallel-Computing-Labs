#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>
//单个block大小
#define THREAD_NUM 256
///矩阵大小
#define MATRIX_SIZE 1000
///block个数
int blocks_num = (MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;

__global__ static void CUDAkernal(const float* a, const float* b, float* c, int n)
{
    //block内的threadID
    const int tid = threadIdx.x;
    //blockID
    const int bid = blockIdx.x;
    //全局threadID
    const int idx = bid * THREAD_NUM + tid;
    const int row = idx / n;
    const int column = idx % n;
    //计算矩阵乘法
    if (row < n && column < n)
    {
        float t = 0;
        for (int i = 0; i < n; i++)
        {
            t += a[row * n + i] * b[i * n + column];
        }
        c[row * n + column] = t;
    }
}

void generateMatrix(float *a, float *b, int n){
    for(int i = 0; i < n * n; i++) {
        a[i] = (float)i / 2;
        b[i] = (float)i / 3;
    }
}

int main() 
{
    //定义矩阵
    float *a, *b, *c;
    int n = MATRIX_SIZE;
    //分配主机端内存
    a = (float*)malloc(sizeof(float)* n * n); 
    b = (float*)malloc(sizeof(float)* n * n); 
    c = (float*)malloc(sizeof(float)* n * n); 

    ///生成矩阵a, b
    generateMatrix(a, b, n);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *cuda_a, *cuda_b, *cuda_c;
    //分配设备端显存 
    hipMalloc((void**)&cuda_a, sizeof(float)* n * n);
    hipMalloc((void**)&cuda_b, sizeof(float)* n * n);
    hipMalloc((void**)&cuda_c, sizeof(float)* n * n);

    //cudaMemcpyHostToDevice - 从内存复制到显存
    //cudaMemcpyDeviceToHost - 从显存复制到内存
    hipMemcpy(cuda_a, a, sizeof(float)* n * n, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(float)* n * n, hipMemcpyHostToDevice);

    ///设备端函数
    CUDAkernal <<< blocks_num, THREAD_NUM, 0 >>>(cuda_a , cuda_b , cuda_c , n);

    //cudaMemcpy 将结果从显存中复制回内存
    hipMemcpy(c, cuda_c, sizeof(float)* n * n, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float timecost;
    hipEventElapsedTime(&timecost, start, stop);
    printf("CUDA time %.4fms\n", timecost);

    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);
    free(a);
    free(b);
    free(c);

    return 0;
}