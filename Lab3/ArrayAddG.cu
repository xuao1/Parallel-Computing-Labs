#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>
//单个block大小
#define THREAD_NUM 256

__global__ void ArrayAdd(int *A, int *B, int *C, int N){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < N) {
        C[index] = A[index] + B[index];
    }
}

int main()
{
    int N;
    scanf("%d", &N);
    int blocks_num = (N + THREAD_NUM - 1) / THREAD_NUM;
    int *A = (int *) malloc(N *sizeof(int));
    int *B = (int *) malloc(N *sizeof(int));
    int *C = (int *) malloc(N *sizeof(int));
    for(int i = 0; i < N; i++){
        A[i] = i;
        B[i] = i;
    }

    int *CUDA_A, *CUDA_B, *CUDA_C;
    hipMalloc(&CUDA_A, N * sizeof(int));
    hipMalloc(&CUDA_B, N * sizeof(int));
    hipMalloc(&CUDA_C, N * sizeof(int));

    hipMemcpy(CUDA_A, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(CUDA_B, B, N * sizeof(int), cudaMemcpyHosttoDevice);

    unsigned int timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer)); //定义计时器
    hipDeviceSynchronize();
    CUT_SAFE_CALL(cutStartTimer(timer)); //计时器启动
    
    ArrayAdd<<<blocks_num, THREAD_NUM, 0>>>(CUDA_A, CUDA_B, CUDA_C, N);

    hipDeviceSynchronize(); //等待计算完成
    CUT_SAFE_CALL(cutStopTimer(timer) ); //计时器停止
    float timecost=cutGetAverageTimerValue(timer); //获得计时结果
    printf("CUDA time %.4fms\n",timecost);
    
    hipFree(CUDA_A);
    hipFree(CUDA_B);
    hipFree(CUDA_C);
    Free(A);
    Free(B);
    Free(C);

    return 0;
}