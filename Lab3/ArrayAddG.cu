#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>
//单个block大小
#define THREAD_NUM 256

__global__ void ArrayAdd(int *A, int *B, int *C, int N){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < N) {
        C[index] = A[index] + B[index];
    }
}

int main()
{
    int N;
    scanf("%d", &N);
    int blocks_num = (N + THREAD_NUM - 1) / THREAD_NUM;
    int *A = (int *) malloc(N *sizeof(int));
    int *B = (int *) malloc(N *sizeof(int));
    int *C = (int *) malloc(N *sizeof(int));
    for(int i = 0; i < N; i++){
        A[i] = i;
        B[i] = i;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int *CUDA_A, *CUDA_B, *CUDA_C;
    hipMalloc(&CUDA_A, N * sizeof(int));
    hipMalloc(&CUDA_B, N * sizeof(int));
    hipMalloc(&CUDA_C, N * sizeof(int));

    hipMemcpy(CUDA_A, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(CUDA_B, B, N * sizeof(int), hipMemcpyHostToDevice);
    
    ArrayAdd<<<blocks_num, THREAD_NUM, 0>>>(CUDA_A, CUDA_B, CUDA_C, N);

    hipMemcpy(C, CUDA_C, sizeof(int) * N, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float timecost;
    hipEventElapsedTime(&timecost, start, stop);
    printf("CUDA time %.4fms\n", timecost);
    
    hipFree(CUDA_A);
    hipFree(CUDA_B);
    hipFree(CUDA_C);
    free(A);
    free(B);
    free(C);

    return 0;
}
